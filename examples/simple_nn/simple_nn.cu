#include "hip/hip_runtime.h"
// nccl_allgather_example.cpp (Data Parallelism)
#include <stdio.h>
#include "nccl.h"
#include "cuda_wrapper.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>


#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}

static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

const double epsilon = 1e-6;

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

__global__ void forward_pass(float *x, float *w, float *y_pred,float N, int n){
  
}

__global__ void backward_pass(float *x, float *w, float *y_pred, float *y_target, float *dw, int n){
  
}


int main(int argc, char* argv[])
{

  int N = 1024; //BatchSize
  int n = 1000; // weight layer
  float lr = 0.01f;

  //MPI initialization
  int myRank, nRanks, localRank = 0;
  
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  // Calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }
  
  ncclUniqueId id;
  ncclComm_t comm;
  hipStream_t s;

  // Get NCCL unique ID at rank 0 and broadcast it to all others
  //if (myRank == 0) {NCCLCHECK(ncclGetUniqueId(&id));}
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));

  // Picking a GPU based on localRank and allocating device buffers
  //CUDACHECK(hipSetDevice(localRank));  
  
  CUDACHECK(hipStreamCreate(&s));
  
  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  // Allocate Host Memory
  float *h_x = (float*)malloc(N * n * sizeof(float));
  float *h_w = (float*)malloc(n*sizeof(float));
  float *h_y_target = (float*)malloc(N*sizeof(float));

  // Initialize input data and weights
  for (int i=0; i<N*n;i++){
    h_x[i] = ((float)rand() / RAND_MAX) *2 - 1;
  }
  for (int i=0; i<n;i++){
    h_w[i] = ((float)rand() / RAND_MAX) * 0.01f;
  }
  for (int i=0; i<N;i++){
    h_y_target[i] = ((float)rand() / RAND_MAX) > 0.5 ? 1.0f : 0.0f;
  }

  //Allocate device memory 
  float *d_x, float *d_w, float *d_y_target, float *d_w_grad, float *d_y_pred;
  CUDACHECK(hipMalloc(&d_x), N*n*sizeof(float)); 
  CUDACHECK(hipMalloc(&d_w), n*sizeof(float)); 
  CUDACHECK(hipMalloc(&d_y_target), N*sizeof(float)); 
  CUDACHECK(hipMalloc(&d_w_grad), n*sizeof(float)); 
  CUDACHECK(hipMalloc(&d_y_pred), N*sizeof(float)); 

  //copy data to device
  CUDACHECK(hipMemcpy(d_x, h_x, N*n*sizeof(float), hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_w, h_w, n*sizeof(float), hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_y_target, h_y_target, N*sizeof(float), hipMemcpyHostToDevice));


  int N_LOOPS = 1;    //No of training loops
  int blockSize = 512;
  int gridSize = (N+blocksize-1) / blockSize;

  for (int tl=0; tl<N_LOOPS; tl++){
    //Zero Gradients
    CUDACHECK(hipMemset(d_w_grad, 0, n*sizeof(float)));  

    //ForwardPass & Synchronize
    forward_pass<<gridSize, blockSize>>(d_x, d_w, d_y_target, N, n);
    CUDACHECK(hipGetLastError());

    //BackwardPass & Synchronize
    backward_pass<<gridSize, blockSize>>(d_x, d_w, d_y_target, d_y_pred, d_w_grad, N, n);
    CUDACHECK(hipGetLastError());


    //Gradient Accumulation using NCCL
    NCCLCHECK(ncclAllReduce((const void*)d_w_grad, (void*)d_w_grad, n, ncclFloat, ncclSum, comm, s)); //SUM
    CUDACHECK(hipStreamSynchronize(s));

    //printf("[MPI Rank %d] Success: ---> gradient content: First = %.2f, Last = %.2f \n", myRank, recvbuff[0], recvbuff[size-1]);

  }

  // Free device buffers
  //CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(d_x));
  CUDACHECK(hipFree(d_w));
  CUDACHECK(hipFree(d_w_grad));
  CUDACHECK(hipFree(d_y_target));
  CUDACHECK(hipFree(d_y_pred));

  free(h_x);
  free(h_w);
  free(h_y_target);

  // Finalize custom NCCL communicator
  NCCLCHECK(ncclCommDestroy(comm));
  
   // Finalizing MPI
  MPICHECK(MPI_Finalize());
  printf("FINALIZED\n");

  return 0;
}

